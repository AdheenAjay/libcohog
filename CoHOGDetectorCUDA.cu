#include "hip/hip_runtime.h"
#include "CoHOGDetectorCUDA.hpp"

void libcohog::set_image(libcohog::gpu_context& context, const unsigned char* ptr, int w, int h)
{
    if(context.w < w)
    {
        if(context.w != 0)
        {
            hipFree(&context.img);
            hipFree(&context.grad);
        }

        hipMalloc(&context.img,  sizeof(unsigned char) * w * h);
        hipMalloc(&context.grad, sizeof(unsigned char) * w * h);
        context.w = w;
        context.h = h;
    }

    hipMemcpy(context.img, ptr, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
}

void libcohog::gpu_context::download()
{
    img_cpu  = new unsigned char[w * h];
    grad_cpu = new unsigned char[w * h];

    hipMemcpy(img_cpu,  img,  sizeof(unsigned char) * w * h, hipMemcpyDeviceToHost);
    hipMemcpy(grad_cpu, grad, sizeof(unsigned char) * w * h, hipMemcpyDeviceToHost);
}


__device__ unsigned char quantitze_gradient(int level, float th, int dx, int dy)
{
    if(dx * dx + dy * dy < th * th)
        return 0xff;

    const float rad     = atan2((float)dy, (float)dx);
    const int   deg     = (int)(rad * 180.0 / M_PI);
    const int   quant   = (int)floor(deg * level / 360.0f + 0.5f);
    const int   norm    = (quant + 2 + level) % level;
    return (unsigned char)norm;
}

__global__ void calc_gradient_kernel(const unsigned char* img, unsigned char* grad, int w, int h, int level, float th)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(w <= x || h <= y)
        return;

    if(x == 0 || y == 0 || w - 1 == x || h - 1 == y)
    {
        grad[y * w + x] = 0xff;
    }

    const int dx =  img[(y + 1) * w + (x + 1)] + img[y * w + (x + 1)] + img[(y - 1) * w + (x + 1)] -
                    img[(y + 1) * w + (x - 1)] - img[y * w + (x - 1)] - img[(y - 1) * w + (x - 1)];
    const int dy =  img[(y - 1) * w + (x - 1)] + img[(y - 1) * w + x] + img[(y - 1) * w + (x + 1)] -
                    img[(y + 1) * w + (x - 1)] - img[(y + 1) * w + x] - img[(y + 1) * w + (x + 1)];

    grad[y * w + x] = quantitze_gradient(level, th, dx, dy);
}

void libcohog::calc_gradient(libcohog::gpu_context& context, int level, float th)
{
    dim3 threads(32, 32);
    dim3 blocks((int)ceil(1.0 * context.w / threads.x), (int)ceil(context.h / threads.y));
    calc_gradient_kernel<<<blocks, threads>>>(context.img, context.grad, context.w, context.h, level, th);
}



